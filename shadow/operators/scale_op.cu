#include "hip/hip_runtime.h"
#include "scale_op.hpp"

namespace Shadow {

namespace Vision {

template <typename T>
__global__ void KernelScale(const T *in_data, int count, const T *scale_data,
                            const T *bias_data, int scale_dim, int inner_dim,
                            T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int index = (globalid / inner_dim) % scale_dim;
    out_data[globalid] =
        in_data[globalid] * scale_data[index] + bias_data[index];
  }
}

template <typename T>
void Scale(const T *in_data, int count, const T *scale_data, const T *bias_data,
           int scale_dim, int inner_dim, T *out_data, Context *context) {
  KernelScale<T><<<GetBlocks(count), NumThreads, 0,
                   hipStream_t(context->hip_stream())>>>(
      in_data, count, scale_data, bias_data, scale_dim, inner_dim, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template void Scale(const float *, int, const float *, const float *, int, int,
                    float *, Context *);

}  // namespace Vision

}  // namespace Shadow
