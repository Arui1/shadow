#include "hip/hip_runtime.h"
#include "image.hpp"
#include "kernel.hpp"
#include "util/log.hpp"

#include "math_functions.h"

namespace Shadow {

namespace Image {

#if defined(USE_CUDA)
template <typename T>
__global__ void KernelDataTransform(const T *in_data, int count, int in_c,
                                    int spatial_dim, float scale, int num_mean,
                                    const T *mean_value, T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int c_out = (globalid / spatial_dim) % in_c;
    int s_out = globalid % spatial_dim;

    if (num_mean == 1) {
      out_data[globalid] = (in_data[globalid] - mean_value[0]) * scale;
    } else if (num_mean == in_c) {
      out_data[globalid] = (in_data[globalid] - mean_value[c_out]) * scale;
    } else if (num_mean == in_c * spatial_dim) {
      out_data[globalid] =
          (in_data[globalid] - mean_value[c_out * spatial_dim + s_out]) * scale;
    }
  }
}

template <typename T>
void DataTransform(const T *in_data, const VecInt &in_shape, float scale,
                   int num_mean, const T *mean_value, T *out_data) {
  int in_c = in_shape[1], spatial_dim = in_shape[2] * in_shape[3];
  int count = in_shape[0] * in_c * spatial_dim;
  KernelDataTransform<T><<<GetBlocks(count), NumThreads>>>(
      in_data, count, in_c, spatial_dim, scale, num_mean, mean_value, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__global__ void KernelIm2Col(const T *in_data, int offset, int count, int in_c,
                             int in_h, int in_w, int kernel_size, int stride,
                             int pad, int dilation, int zero_point, int out_h,
                             int out_w, T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int h_index = globalid / out_w;
    int h_col = h_index % out_h;
    int w_col = globalid % out_w;
    int c_im = h_index / out_h;
    int c_col = c_im * kernel_size * kernel_size;
    int h_offset = h_col * stride - pad;
    int w_offset = w_col * stride - pad;
    out_data += (c_col * out_h + h_col) * out_w + w_col;
    in_data += offset + (c_im * in_h + h_offset) * in_w + w_offset;
    for (int i = 0; i < kernel_size; ++i) {
      for (int j = 0; j < kernel_size; ++j) {
        int h_im = h_offset + i * dilation;
        int w_im = w_offset + j * dilation;
        *out_data = (h_im >= 0 && w_im >= 0 && h_im < in_h && w_im < in_w)
                        ? in_data[i * dilation * in_w + j * dilation]
                        : static_cast<T>(zero_point);
        out_data += out_h * out_w;
      }
    }
  }
}

template <typename T>
void Im2Col(const T *in_data, const VecInt &in_shape, int offset,
            int kernel_size, int stride, int pad, int dilation, int zero_point,
            const VecInt &out_shape, T *out_data) {
  int in_c = in_shape[1], in_h = in_shape[2], in_w = in_shape[3];
  int out_h = out_shape[2], out_w = out_shape[3];
  int count = in_c * out_h * out_w;
  KernelIm2Col<T><<<GetBlocks(count), NumThreads>>>(
      in_data, offset, count, in_c, in_h, in_w, kernel_size, stride, pad,
      dilation, zero_point, out_h, out_w, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__global__ void KernelPooling(const T *in_data, int count, int in_c, int in_h,
                              int in_w, int kernel_size, int stride, int pad,
                              int mode, int out_h, int out_w, T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int temp = globalid / out_w;
    int j_out = globalid % out_w;
    int i_out = temp % out_h;
    temp = temp / out_h;
    int c_out = temp % in_c;
    int b_out = temp / in_c;

    int kistart = i_out * stride - pad, kjstart = j_out * stride - pad;
    int kiend = min(kistart + kernel_size, in_h);
    int kjend = min(kjstart + kernel_size, in_w);
    int pool_size = (kiend - kistart) * (kjend - kjstart);
    kistart = max(kistart, 0), kjstart = max(kjstart, 0);
    kiend = min(kiend, in_h), kjend = min(kjend, in_w);

    T max = -FLT_MAX;
    T sum = T(0);
    for (int ki = kistart; ki < kiend; ++ki) {
      for (int kj = kjstart; kj < kjend; ++kj) {
        int index = kj + in_w * (ki + in_h * (c_out + in_c * b_out));
        T value = in_data[index];
        max = (value > max) ? value : max;
        sum += value;
      }
    }
    out_data[globalid] = (mode == 0) ? max : sum / pool_size;
  }
}

template <typename T>
void Pooling(const T *in_data, const VecInt &in_shape, int kernel_size,
             int stride, int pad, int mode, const VecInt &out_shape,
             T *out_data) {
  int batch = in_shape[0];
  int in_c = in_shape[1], in_h = in_shape[2], in_w = in_shape[3];
  int out_h = out_shape[2], out_w = out_shape[3];
  int count = batch * in_c * out_h * out_w;
  KernelPooling<T><<<GetBlocks(count), NumThreads>>>(
      in_data, count, in_c, in_h, in_w, kernel_size, stride, pad, mode, out_h,
      out_w, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__global__ void KernelConcat(const T *in_data, int count, int num_concats,
                             int concat_size, int top_concat_axis,
                             int bottom_concat_axis, int offset_concat_axis,
                             T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int total_concat_size = concat_size * bottom_concat_axis;
    int concat_num = globalid / total_concat_size;
    int concat_index = globalid % total_concat_size;
    int top_index =
        concat_index +
        (concat_num * top_concat_axis + offset_concat_axis) * concat_size;
    out_data[top_index] = in_data[globalid];
  }
}

template <typename T>
void Concat(const T *in_data, int count, int num_concats, int concat_size,
            int top_concat_axis, int bottom_concat_axis, int offset_concat_axis,
            T *out_data) {
  KernelConcat<T><<<GetBlocks(count), NumThreads>>>(
      in_data, count, num_concats, concat_size, top_concat_axis,
      bottom_concat_axis, offset_concat_axis, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__global__ void KernelPermute(const T *in_data, int count, int num_axes,
                              const int *permute_order, const int *old_steps,
                              const int *new_steps, T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int old_idx = 0;
    int idx = globalid;
    for (int j = 0; j < num_axes; ++j) {
      int order = permute_order[j];
      old_idx += (idx / new_steps[j]) * old_steps[order];
      idx %= new_steps[j];
    }
    out_data[globalid] = in_data[old_idx];
  }
}

template <typename T, typename Dtype>
void Permute(const T *in_data, int count, int num_axes,
             const Dtype *permute_order, const Dtype *old_steps,
             const Dtype *new_steps, T *out_data) {
  KernelPermute<T><<<GetBlocks(count), NumThreads>>>(
      in_data, count, num_axes, permute_order, old_steps, new_steps, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__global__ void KernelScale(const T *in_data, int count, const T *scale_data,
                            const T *bias_data, int scale_dim, int inner_dim,
                            T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int index = (globalid / inner_dim) % scale_dim;
    out_data[globalid] =
        in_data[globalid] * scale_data[index] + bias_data[index];
  }
}

template <typename T>
void Scale(const T *in_data, int count, const T *scale_data, const T *bias_data,
           int scale_dim, int inner_dim, T *out_data) {
  KernelScale<T><<<GetBlocks(count), NumThreads>>>(
      in_data, count, scale_data, bias_data, scale_dim, inner_dim, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__global__ void KernelBias(const T *in_data, int count, const T *bias_data,
                           int bias_dim, int inner_dim, T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int index = (globalid / inner_dim) % bias_dim;
    out_data[globalid] = in_data[globalid] + bias_data[index];
  }
}

template <typename T>
void Bias(const T *in_data, int count, const T *bias_data, int bias_dim,
          int inner_dim, T *out_data) {
  KernelBias<T><<<GetBlocks(count), NumThreads>>>(
      in_data, count, bias_data, bias_dim, inner_dim, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__global__ void KernelReorg(const T *in_data, int count, int in_c, int in_h,
                            int in_w, int out_c, int out_h, int out_w,
                            int stride, T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int temp = globalid / out_w;
    int w = globalid % out_w;
    int h = temp % out_h;
    temp = temp / out_h;
    int c = temp % out_c;
    int b = temp / out_c;

    int c_in = c % in_c;
    int area = c / in_c;
    int h_in = h * stride + area / stride;
    int w_in = w * stride + area % stride;
    int in_index = ((b * in_c + c_in) * in_h + h_in) * in_w + w_in;
    int out_index = ((b * out_c + c) * out_h + h) * out_w + w;
    out_data[out_index] = in_data[in_index];
  }
}

template <typename T>
void Reorg(const T *in_data, const VecInt &in_shape, int stride, T *out_data) {
  int batch = in_shape[0];
  int in_c = in_shape[1], in_h = in_shape[2], in_w = in_shape[3];
  int out_c = in_c * stride * stride;
  int out_h = in_h / stride, out_w = in_w / stride;
  int count = batch * out_c * out_h * out_w;
  KernelReorg<T><<<GetBlocks(count), NumThreads>>>(
      in_data, count, in_c, in_h, in_w, out_c, out_h, out_w, stride, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__global__ void KernelLRNFillScale(const T *in_data, int count, int in_c,
                                   int in_h, int in_w, int size,
                                   float alpha_over_size, float k,
                                   T *scale_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int temp = globalid / in_w;
    int w = globalid % in_w;
    int h = temp % in_h;
    int b = temp / in_h;

    int offset = (b * in_c * in_h + h) * in_w + w, head = 0;
    const T *in_off = in_data + offset;
    T *scale_off = scale_data + offset;
    T accum_scale = T(0);
    int step = in_h * in_w;
    int pre_pad = (size - 1) / 2, post_pad = size - pre_pad - 1;
    while (head < post_pad && head < in_c) {
      accum_scale += in_off[head * step] * in_off[head * step];
      head++;
    }
    while (head < in_c) {
      accum_scale += in_off[head * step] * in_off[head * step];
      if (head - size >= 0) {
        accum_scale -=
            in_off[(head - size) * step] * in_off[(head - size) * step];
      }
      scale_off[(head - post_pad) * step] = k + accum_scale * alpha_over_size;
      head++;
    }
    while (head < in_c + post_pad) {
      if (head - size >= 0) {
        accum_scale -=
            in_off[(head - size) * step] * in_off[(head - size) * step];
      }
      scale_off[(head - post_pad) * step] = k + accum_scale * alpha_over_size;
      head++;
    }
  }
}

template <typename T>
__global__ void KernelLRN(const T *in_data, int count, const T *scale_data,
                          float negative_beta, T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    out_data[globalid] =
        in_data[globalid] * pow(scale_data[globalid], negative_beta);
  }
}

template <typename T>
void LRN(const T *in_data, const VecInt &in_shape, int size, float alpha,
         float beta, float k, T *scale_data, T *out_data) {
  int batch = in_shape[0], in_c = in_shape[1];
  int in_h = in_shape[2], in_w = in_shape[3];
  int count = batch * in_h * in_w;
  KernelLRNFillScale<T><<<GetBlocks(count), NumThreads>>>(
      in_data, count, in_c, in_h, in_w, size, alpha / size, k, scale_data);
  CUDA_CHECK(hipPeekAtLastError());
  count *= in_c;
  KernelLRN<T><<<GetBlocks(count), NumThreads>>>(in_data, count, scale_data,
                                                 -beta, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__device__ float ActivateValue(T x, int type) {
  switch (type) {
    case 1:
      return x * (x > 0); /*relu*/
    case 2:
      return (x > 0) ? x : T(.1) * x; /*leaky*/
    default:
      return x;
  }
}

template <typename T>
__global__ void KernelActivate(T *data, int count, int type) {
  CUDA_KERNEL_LOOP(globalid, count) {
    data[globalid] = ActivateValue(data[globalid], type);
  }
}

template <typename T>
void Activate(T *data, int count, int type) {
  KernelActivate<T><<<GetBlocks(count), NumThreads>>>(data, count, type);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
__global__ void KernelPRelu(T *data, int count, int channels, int dim,
                            int div_factor, const T *slope_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int c = (globalid / dim) % channels / div_factor;
    T value = data[globalid];
    data[globalid] = value > 0 ? value : value * slope_data[c];
  }
}

template <typename T>
void PRelu(T *data, const VecInt &in_shape, bool channel_shared,
           const T *slope_data) {
  int channels = in_shape[1], dim = 1;
  for (int i = 2; i < in_shape.size(); ++i) dim *= in_shape[i];
  int count = in_shape[0] * channels * dim;
  int div_factor = channel_shared ? channels : 1;
  KernelPRelu<T><<<GetBlocks(count), NumThreads>>>(data, count, channels, dim,
                                                   div_factor, slope_data);
  CUDA_CHECK(hipPeekAtLastError());
}

// Explicit instantiation
template void DataTransform(const float *in_data, const VecInt &in_shape,
                            float scale, int num_mean, const float *mean_value,
                            float *out_data);
template void Im2Col(const float *in_data, const VecInt &in_shape, int offset,
                     int kernel_size, int stride, int pad, int dilation,
                     int zero_point, const VecInt &out_shape, float *out_data);
template void Pooling(const float *in_data, const VecInt &in_shape,
                      int kernel_size, int stride, int pad, int mode,
                      const VecInt &out_shape, float *out_data);
template void Concat(const float *in_data, int count, int num_concats,
                     int concat_size, int top_concat_axis,
                     int bottom_concat_axis, int offset_concat_axis,
                     float *out_data);
template void Permute(const float *in_data, int count, int num_axes,
                      const int *permute_order, const int *old_steps,
                      const int *new_steps, float *out_data);
template void Scale(const float *in_data, int count, const float *scale_data,
                    const float *bias_data, int scale_dim, int inner_dim,
                    float *out_data);
template void Bias(const float *in_data, int count, const float *bias_data,
                   int bias_dim, int inner_dim, float *out_data);
template void Reorg(const float *in_data, const VecInt &in_shape, int stride,
                    float *out_data);
template void LRN(const float *in_data, const VecInt &in_shape, int size,
                  float alpha, float beta, float k, float *scale_data,
                  float *out_data);
template void Activate(float *data, int count, int type);
template void PRelu(float *data, const VecInt &in_shape, bool channel_shared,
                    const float *slope_data);
#endif

}  // namespace Image

}  // namespace Shadow
