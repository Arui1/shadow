#include "allocator.hpp"

#include "common.hpp"

namespace Shadow {

#if defined(USE_CUDA)

class GPUAllocator : public Allocator {
 public:
  Device GetDevice() const override { return Device::kGPU; }

  void *MakeBuffer(size_t size, const void *host_ptr) const override {
    void *ptr;
    CUDA_CHECK(hipMalloc(&ptr, size));
    if (host_ptr != nullptr) {
      WriteBuffer(size, host_ptr, ptr);
    }
    return ptr;
  }

  void ReadBuffer(size_t size, const void *src, void *dst) const override {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
  }

  void WriteBuffer(size_t size, const void *src, void *dst) const override {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
  }

  void CopyBuffer(size_t size, const void *src, void *dst) const override {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
  }

  void ReleaseBuffer(void *ptr) const override { CUDA_CHECK(hipFree(ptr)); }
};

template <>
Allocator *GetAllocator<Device::kGPU>() {
  static GPUAllocator allocator;
  return &allocator;
}

#endif

}  // namespace Shadow
