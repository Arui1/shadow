#include "hip/hip_runtime.h"
#include "shadow/kernel.hpp"

__global__ void DataTransformKernel(int N, const float *in_data, float scale,
                                    float mean_value, float *out_data) {
  int globalid =
      (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (globalid > N)
    return;

  out_data[globalid] = (in_data[globalid] - mean_value) * scale;
}

void Kernel::DataTransform(int N, const float *in_data, float scale,
                           float mean_value, float *out_data) {
  DataTransformKernel<<<CUDA::CUDAGridDim(N), BLOCK>>>(N, in_data, scale,
                                                       mean_value, out_data);
  CUDA::CUDACheckError(hipPeekAtLastError());
}

__global__ void Im2ColKernel(const float *im_data, int offset, int in_c,
                             int in_h, int in_w, int ksize, int stride, int pad,
                             int out_h, int out_w, float *col_data) {
  int globalid =
      (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (globalid >= in_c * out_h * out_w)
    return;

  int c_out = (globalid / (out_w * out_h)) % in_c;
  int i_out = (globalid / out_w) % out_h;
  int j_out = globalid % out_w;

  int i_inp = -pad + i_out * stride;
  int j_inp = -pad + j_out * stride;

  im_data += offset + c_out * in_h * in_w;
  col_data += (c_out * ksize * ksize * out_h + i_out) * out_w + j_out;

  for (int ki = 0; ki < ksize; ++ki) {
    for (int kj = 0; kj < ksize; ++kj) {
      int i = i_inp + ki;
      int j = j_inp + kj;
      *col_data = (i >= 0 && j >= 0 && i < in_h && j < in_w)
                      ? im_data[i * in_w + j]
                      : 0.f;
      col_data += out_h * out_w;
    }
  }
}

void Kernel::Im2Col(const float *im_data, int offset, int in_c, int in_h,
                    int in_w, int ksize, int stride, int pad, int out_h,
                    int out_w, float *col_data) {
  int N = in_c * out_h * out_w;
  Im2ColKernel<<<CUDA::CUDAGridDim(N), BLOCK>>>(im_data, offset, in_c, in_h,
                                                in_w, ksize, stride, pad, out_h,
                                                out_w, col_data);
  CUDA::CUDACheckError(hipPeekAtLastError());
}

__global__ void PoolingKernel(const float *in_data, int batch, int in_c,
                              int in_h, int in_w, int ksize, int stride,
                              int out_h, int out_w, int mode, float *out_data) {
  int globalid =
      (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (globalid >= batch * in_c * out_h * out_w)
    return;

  int h_offset = ((in_h - ksize) % stride) / 2;
  int w_offset = ((in_w - ksize) % stride) / 2;

  int b_out = (globalid / (out_w * out_h * in_c)) % batch;
  int c_out = (globalid / (out_w * out_h)) % in_c;
  int i_out = (globalid / out_w) % out_h;
  int j_out = globalid % out_w;

  int i_inp = h_offset + i_out * stride;
  int j_inp = w_offset + j_out * stride;

  int offset = ((b_out * in_c + c_out) * in_h + i_inp) * in_w + j_inp;

  float max = -10000.0f;
  float sum = 0.f;
  for (int ki = 0; ki < ksize; ++ki) {
    for (int kj = 0; kj < ksize; ++kj) {
      int in = offset + ki * in_w + kj;
      bool valid = in < batch * in_c * in_h * in_w;
      float value = valid ? in_data[in] : -10000.0f;
      max = (value > max) ? value : max;
      sum += valid ? in_data[in] : 0.f;
    }
  }
  if (mode == 0)
    out_data[globalid] = max;
  else
    out_data[globalid] = sum / (ksize * ksize);
}

void Kernel::Pooling(const float *in_data, int batch, int in_c, int in_h,
                     int in_w, int ksize, int stride, int out_h, int out_w,
                     int mode, float *out_data) {
  int N = batch * in_c * out_h * out_w;
  PoolingKernel<<<CUDA::CUDAGridDim(N), BLOCK>>>(in_data, batch, in_c, in_h,
                                                 in_w, ksize, stride, out_h,
                                                 out_w, mode, out_data);
  CUDA::CUDACheckError(hipPeekAtLastError());
}

__device__ float Activate(float x, int mode) {
  switch (mode) {
  case 0:
    return x; /*linear*/
  case 1:
    return x * (x > 0); /*relu*/
  case 2:
    return (x > 0) ? x : .1f * x; /*leaky*/
  default:
    return x;
  }
}

__global__ void ActivateArrayKernel(int N, int mode, float *out_data) {
  int globalid =
      (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (globalid >= N)
    return;

  out_data[globalid] = Activate(out_data[globalid], mode);
}

void Kernel::ActivateArray(int N, shadow::ActivateType a, float *out_data) {
  ActivateArrayKernel<<<CUDA::CUDAGridDim(N), BLOCK>>>(N, a, out_data);
  CUDA::CUDACheckError(hipPeekAtLastError());
}

__global__ void SetArrayKernel(int N, float value, float *out_data) {
  int globalid =
      (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (globalid >= N)
    return;

  out_data[globalid] = value;
}

void Kernel::SetArray(int N, float value, float *out_data) {
  float val = {value};
  SetArrayKernel<<<CUDA::CUDAGridDim(N), BLOCK>>>(N, val, out_data);
  CUDA::CUDACheckError(hipPeekAtLastError());
}

__global__ void SetArrayRepeatKernel(int N, const float *value, int value_size,
                                     float *out_data) {
  int globalid =
      (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (globalid >= N * value_size)
    return;

  int value_index = globalid / N;
  out_data[globalid] = value[value_index];
}

void Kernel::SetArrayRepeat(int N, const float *value, int value_size,
                            float *out_data) {
  SetArrayRepeatKernel<<<CUDA::CUDAGridDim(N * value_size), BLOCK>>>(
      N, value, value_size, out_data);
  CUDA::CUDACheckError(hipPeekAtLastError());
}
