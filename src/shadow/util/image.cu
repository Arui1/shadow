#include "hip/hip_runtime.h"
#include "shadow/kernel.hpp"
#include "shadow/util/image.hpp"

namespace Image {

#if defined(USE_CUDA)
__global__ void KernelDataTransform(const float *in_data, int count, int in_c,
                                    int spatial_dim, float scale, int num_mean,
                                    const float *mean_value, float *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int c_out = (globalid / spatial_dim) % in_c;
    int s_out = globalid % spatial_dim;

    if (num_mean == 1) {
      out_data[globalid] = (in_data[globalid] - mean_value[0]) * scale;
    } else if (num_mean == in_c) {
      out_data[globalid] = (in_data[globalid] - mean_value[c_out]) * scale;
    } else if (num_mean == in_c * spatial_dim) {
      out_data[globalid] =
          (in_data[globalid] - mean_value[c_out * spatial_dim + s_out]) * scale;
    }
  }
}

template <typename T>
void DataTransform(const T *in_data, const VecInt &in_shape, float scale,
                   int num_mean, const T *mean_value, T *out_data) {
  int in_c = in_shape[1], spatial_dim = in_shape[2] * in_shape[3];
  int count = in_shape[0] * in_c * spatial_dim;
  KernelDataTransform<<<GetBlocks(count), NumThreads>>>(
      in_data, count, in_c, spatial_dim, scale, num_mean, mean_value, out_data);
  CheckError(hipPeekAtLastError());
}

__global__ void KernelIm2Col(const float *im_data, int offset, int in_c,
                             int in_h, int in_w, int kernel_size, int stride,
                             int pad, int dilation, int out_h, int out_w,
                             float *col_data) {
  CUDA_KERNEL_LOOP(globalid, in_c * out_h * out_w) {
    const int h_index = globalid / out_w;
    const int h_col = h_index % out_h;
    const int w_col = globalid % out_w;
    const int c_im = h_index / out_h;
    const int c_col = c_im * kernel_size * kernel_size;
    const int h_offset = h_col * stride - pad;
    const int w_offset = w_col * stride - pad;
    col_data += (c_col * out_h + h_col) * out_w + w_col;
    im_data += offset + (c_im * in_h + h_offset) * in_w + w_offset;
    for (int i = 0; i < kernel_size; ++i) {
      for (int j = 0; j < kernel_size; ++j) {
        int h_im = h_offset + i * dilation;
        int w_im = w_offset + j * dilation;
        *col_data = (h_im >= 0 && w_im >= 0 && h_im < in_h && w_im < in_w)
                        ? im_data[i * dilation * in_w + j * dilation]
                        : 0;
        col_data += out_h * out_w;
      }
    }
  }
}

template <typename T>
void Im2Col(const T *in_data, const VecInt &in_shape, int offset,
            int kernel_size, int stride, int pad, int dilation,
            const VecInt &out_shape, T *out_data) {
  int in_c = in_shape[1], in_h = in_shape[2], in_w = in_shape[3];
  int out_h = out_shape[2], out_w = out_shape[3];
  int N = in_c * out_h * out_w;
  KernelIm2Col<<<GetBlocks(N), NumThreads>>>(in_data, offset, in_c, in_h, in_w,
                                             kernel_size, stride, pad, dilation,
                                             out_h, out_w, out_data);
  CheckError(hipPeekAtLastError());
}

__global__ void KernelPooling(const float *in_data, int batch, int in_c,
                              int in_h, int in_w, int kernel_size, int stride,
                              int pad, int mode, int out_h, int out_w,
                              float *out_data) {
  CUDA_KERNEL_LOOP(globalid, batch * in_c * out_h * out_w) {
    int b_out = (globalid / (out_w * out_h * in_c)) % batch;
    int c_out = (globalid / (out_w * out_h)) % in_c;
    int i_out = (globalid / out_w) % out_h;
    int j_out = globalid % out_w;

    int kistart = i_out * stride - pad, kjstart = j_out * stride - pad;
    int kiend = min(kistart + kernel_size, in_h);
    int kjend = min(kjstart + kernel_size, in_w);
    int pool_size = (kiend - kistart) * (kjend - kjstart);
    kistart = max(kistart, 0), kjstart = max(kjstart, 0);
    kiend = min(kiend, in_h), kjend = min(kjend, in_w);

    float max = -FLT_MAX;
    float sum = 0.f;
    for (int ki = kistart; ki < kiend; ++ki) {
      for (int kj = kjstart; kj < kjend; ++kj) {
        int index = kj + in_w * (ki + in_h * (c_out + in_c * b_out));
        float value = in_data[index];
        max = (value > max) ? value : max;
        sum += value;
      }
    }
    if (mode == 0) {
      out_data[globalid] = max;
    } else {
      out_data[globalid] = sum / pool_size;
    }
  }
}

template <typename T>
void Pooling(const T *in_data, const VecInt &in_shape, int kernel_size,
             int stride, int pad, int mode, const VecInt &out_shape,
             T *out_data) {
  int batch = in_shape[0];
  int in_c = in_shape[1], in_h = in_shape[2], in_w = in_shape[3];
  int out_h = out_shape[2], out_w = out_shape[3];
  int N = batch * in_c * out_h * out_w;
  KernelPooling<<<GetBlocks(N), NumThreads>>>(in_data, batch, in_c, in_h, in_w,
                                              kernel_size, stride, pad, mode,
                                              out_h, out_w, out_data);
  CheckError(hipPeekAtLastError());
}

__global__ void KernelConcat(const float *in_data, int count, int num_concats,
                             int concat_size, int top_concat_axis,
                             int bottom_concat_axis, int offset_concat_axis,
                             float *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int total_concat_size = concat_size * bottom_concat_axis;
    int concat_num = globalid / total_concat_size;
    int concat_index = globalid % total_concat_size;
    int top_index =
        concat_index +
        (concat_num * top_concat_axis + offset_concat_axis) * concat_size;
    out_data[top_index] = in_data[globalid];
  }
}

template <typename T>
void Concat(const T *in_data, int count, int num_concats, int concat_size,
            int top_concat_axis, int bottom_concat_axis, int offset_concat_axis,
            T *out_data) {
  KernelConcat<<<GetBlocks(count), NumThreads>>>(
      in_data, count, num_concats, concat_size, top_concat_axis,
      bottom_concat_axis, offset_concat_axis, out_data);
  CheckError(hipPeekAtLastError());
}

__global__ void KernelPermute(const float *in_data, int count, int num_axes,
                              const int *permute_order, const int *old_steps,
                              const int *new_steps, float *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int old_idx = 0;
    int idx = globalid;
    for (int j = 0; j < num_axes; ++j) {
      int order = permute_order[j];
      old_idx += (idx / new_steps[j]) * old_steps[order];
      idx %= new_steps[j];
    }
    out_data[globalid] = in_data[old_idx];
  }
}

template <typename T, typename Dtype>
void Permute(const T *in_data, int count, int num_axes,
             const Dtype *permute_order, const Dtype *old_steps,
             const Dtype *new_steps, T *out_data) {
  KernelPermute<<<GetBlocks(count), NumThreads>>>(
      in_data, count, num_axes, permute_order, old_steps, new_steps, out_data);
  CheckError(hipPeekAtLastError());
}

__device__ float ActivateValue(float x, int type) {
  switch (type) {
    case 0:
      return x; /*linear*/
    case 1:
      return x * (x > 0); /*relu*/
    case 2:
      return (x > 0) ? x : .1f * x; /*leaky*/
    default:
      return x;
  }
}

__global__ void KernelActivate(float *data, int count, int type) {
  CUDA_KERNEL_LOOP(globalid, count) {
    data[globalid] = ActivateValue(data[globalid], type);
  }
}

template <typename T>
void Activate(T *data, int count, int type) {
  KernelActivate<<<GetBlocks(count), NumThreads>>>(data, count, type);
  CheckError(hipPeekAtLastError());
}

// Explicit instantiation
template void DataTransform(const float *in_data, const VecInt &in_shape,
                            float scale, int num_mean, const float *mean_value,
                            float *out_data);
template void Im2Col(const float *in_data, const VecInt &in_shape, int offset,
                     int kernel_size, int stride, int pad, int dilation,
                     const VecInt &out_shape, float *out_data);
template void Pooling(const float *in_data, const VecInt &in_shape,
                      int kernel_size, int stride, int pad, int mode,
                      const VecInt &out_shape, float *out_data);
template void Concat(const float *in_data, int count, int num_concats,
                     int concat_size, int top_concat_axis,
                     int bottom_concat_axis, int offset_concat_axis,
                     float *out_data);
template void Permute(const float *in_data, int count, int num_axes,
                      const int *permute_order, const int *old_steps,
                      const int *new_steps, float *out_data);
template void Activate(float *data, int count, int type);
#endif

}  // namespace Image
